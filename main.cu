#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <thread>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include <GLFW/glfw3.h>
#include "include/cuda/vec3.h"
#include "include/cuda/ray.h"
#include "include/cuda/sphere.h"
#include "include/cuda/hitable_list.h"
#include "include/cuda/camera.h"
#include "include/cuda/material.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable **d_list, hitable **d_world, hiprandState *rand_state, int num_spheres) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;

        // Ground sphere
        d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));

        // Add user-defined spheres
        for (int i = 1; i <= num_spheres; ++i) {
            float choose_mat = RND;
            vec3 center(RND * 10 - 5, 0.2, RND * 10 - 5);
            if (choose_mat < 0.8f) {
                d_list[i] = new sphere(vec3(RND * 10 - 5, 0.8, RND * 10 - 5), 0.8, new lambertian(vec3(RND * RND, RND * RND, RND * RND)));
            } else if (choose_mat < 0.95f) {
                d_list[i] = new sphere(center, 0.4, new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND));
            } else {
                d_list[i] = new sphere(center, 0.2, new dielectric(1.5));
            }
        }

        *rand_state = local_rand_state;
        *d_world = new hitable_list(d_list, num_spheres + 1);
    }
}

__global__ void create_camera(camera **d_camera, int nx, int ny, float angle) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        vec3 lookfrom(13 * cos(angle), 2, 13 * sin(angle));
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0; // (lookfrom - lookat).length();
        float aperture = 0.1;
        *d_camera = new camera(lookfrom, lookat, vec3(0, 1, 0), 30.0, float(nx) / float(ny), aperture, dist_to_focus);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera, int num_spheres) {
    for(int i=0; i < num_spheres + 1; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

void update_texture(unsigned char* image, vec3 *fb, int width, int height) {
    for (int j = 0; j < height; ++j) {
        for (int i = 0; i < width; ++i) {
            size_t pixel_index = j * width + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            image[3 * pixel_index + 0] = ir;
            image[3 * pixel_index + 1] = ig;
            image[3 * pixel_index + 2] = ib;
        }
    }
}

int main() {
    int nx = 1200;
    int ny = 800;
    int ns = 10;
    int tx = 8;
    int ty = 8;
    int num_spheres;

    // User input for number of spheres
    std::cout << "Enter the number of spheres to render: ";
    std::cin >> num_spheres;

    // User input for frames per second
    int fps;
    std::cout << "Enter the desired frames per second: ";
    std::cin >> fps;

    if (!glfwInit()) {
        std::cerr << "Failed to initialize GLFW" << std::endl;
        return -1;
    }

    GLFWwindow* window = glfwCreateWindow(nx, ny, "CUDA Path Tracer", NULL, NULL);
    if (!window) {
        std::cerr << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);
    glEnable(GL_TEXTURE_2D);

    GLuint texture;
    glGenTextures(1, &texture);
    glBindTexture(GL_TEXTURE_2D, texture);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    unsigned char* image = new unsigned char[nx * ny * 3];

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1, 1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // allocate memory for world and camera
    hitable** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, (num_spheres + 1) * sizeof(hitable*))); // +1 for the ground sphere
    hitable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));

    // Create our world of hitables once
    create_world<<<1, 1>>>(d_list, d_world, d_rand_state2, num_spheres);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto frame_duration = std::chrono::milliseconds(1000 / fps);

    for (int frame = 0; !glfwWindowShouldClose(window); ++frame) {
        auto frame_start = std::chrono::high_resolution_clock::now();
        float angle = 2.0f * M_PI * frame / fps;

        // Update camera position
        create_camera<<<1, 1>>>(d_camera, nx, ny, angle);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        clock_t start, stop;
        start = clock();
        // Render our buffer
        dim3 blocks(nx / tx + 1, ny / ty + 1);
        dim3 threads(tx, ty);
        render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        stop = clock();
        double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
        std::cerr << "Frame " << frame << " took " << timer_seconds << " seconds.\n";

        // Update OpenGL texture
        update_texture(image, fb, nx, ny);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, nx, ny, 0, GL_RGB, GL_UNSIGNED_BYTE, image);
        glClear(GL_COLOR_BUFFER_BIT);
        glBegin(GL_QUADS);
        glTexCoord2f(0.0f, 0.0f);
        glVertex2f(-1.0f, -1.0f);
        glTexCoord2f(1.0f, 0.0f);
        glVertex2f(1.0f, -1.0f);
        glTexCoord2f(1.0f, 1.0f);
        glVertex2f(1.0f, 1.0f);
        glTexCoord2f(0.0f, 1.0f);
        glVertex2f(-1.0f, 1.0f);
        glEnd();
        glfwSwapBuffers(window);
        glfwPollEvents();

        auto frame_end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> frame_time = frame_end - frame_start;
        if (frame_time < frame_duration) {
            std::this_thread::sleep_for(frame_duration - frame_time);
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());

    // Free world objects
    free_world<<<1, 1>>>(d_list, d_world, d_camera, num_spheres);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    delete[] image;
    glfwDestroyWindow(window);
    glfwTerminate();

    hipDeviceReset();
    return 0;
}
